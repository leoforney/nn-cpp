#include "hip/hip_runtime.h"
#include "Dense.h"

#include <iostream>

#include "../Types.h"
#include "../Utils.h"
#include "Layer.h"
#include "thrust/host_vector.h"
#include "thrust/device_vector.h"

__global__ void matmulKernel(float *A, float *B, float *C, int M, int K, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row < M && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < K; ++k) {
            sum += A[row * K + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

void computeMatMul(const thrust::device_vector<float>& A, const thrust::device_vector<float>& B, thrust::device_vector<float>& C, int M, int K, int N) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    matmulKernel<<<blocksPerGrid, threadsPerBlock>>>(thrust::raw_pointer_cast(A.data()),
                                                     thrust::raw_pointer_cast(B.data()),
                                                     thrust::raw_pointer_cast(C.data()),
                                                     M, K, N);

    hipDeviceSynchronize();
}

namespace ML {
    void DenseLayer::computeNaive(const LayerData& dataIn) const {
        Array1D_fp32 dataInArray = dataIn.getData<Array1D_fp32>();
        std::size_t inputSize = getInputParams().dims[0];

        Array1D_fp32 dataOutArray = outData.getData<Array1D_fp32>();
        std::size_t outSize = getOutputParams().dims[0];

        Array1D<fp32> biasDataArray = biasData.getData<Array1D<fp32>>();
        std::size_t biasSize = biasParam.dims[0];

        Array2D<fp32> weightDataArray = weightData.getData<Array2D<fp32>>();
        std::size_t weightDimX = weightParam.dims[0];
        std::size_t weightDimY = weightParam.dims[1];

        if (inputSize != weightDimY || outSize != weightDimX || outSize != biasSize) {
            throw std::invalid_argument("Dimension mismatch in DenseLayer::computeNaive");
        }

        for (std::size_t i = 0; i < outSize; ++i) {
            dataOutArray[i] = 0.0f;
        }

        for (std::size_t i = 0; i < outSize; ++i) {
            for (std::size_t j = 0; j < inputSize; ++j) {
                dataOutArray[i] += dataInArray[j] * weightDataArray[i][j];
            }
        }

        for (std::size_t i = 0; i < outSize; ++i) {
            dataOutArray[i] += biasDataArray[i];
        }
    }

    void DenseLayer::computeSIMD(const LayerData& dataIn) const {
        std::size_t M = 1;
        std::size_t K = getInputParams().dims[0];
        std::size_t N = getOutputParams().dims[0];

        thrust::device_vector<float> d_A(dataIn.getData<Array1D_fp32>(), dataIn.getData<Array1D_fp32>() + K);
        thrust::device_vector<float> d_B(weightData.getData<Array2D_fp32>(), weightData.getData<Array2D_fp32>() + K * N);
        thrust::device_vector<float> d_C(N);
        thrust::device_vector<float> d_bias(biasData.getData<Array1D<fp32>>(), biasData.getData<Array1D<fp32>>() + N);

        computeMatMul(d_A, d_B, d_C, M, K, N);

        for (size_t i = 0; i < N; ++i) {
            thrust::transform(d_C.begin() + i, d_C.begin() + i + 1, d_bias.begin() + i, d_C.begin() + i, thrust::plus<float>());
        }

        thrust::copy(d_C.begin(), d_C.end(), outData.getData<Array1D_fp32>());
    }

};  // namespace ML