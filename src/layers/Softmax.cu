#include "hip/hip_runtime.h"
#include "Softmax.h"

#include <iostream>

#include "../Types.h"
#include "../Utils.h"
#include "Layer.h"

namespace ML {

    void SoftmaxLayer::computeNaive(const LayerData& dataIn) const {
        Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
        std::size_t inputHeight = getInputParams().dims[0];
        std::size_t inputWidth = getInputParams().dims[1];
        std::size_t inputColorChannels = getInputParams().dims[2];

        Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();
        auto outDims = getOutputParams();
        std::size_t outHeight = outDims.dims[0];
        std::size_t outWidth = outDims.dims[1];
        std::size_t outColorChannels = outDims.dims[2];

        for (std::size_t i = 0; i < outHeight; ++i) {
            for (std::size_t j = 0; j < outWidth; ++j) {
                for (std::size_t c = 0; c < outColorChannels; ++c) {
                    float exp_val = std::exp(dataIn3DArray[i][j][c]);
                    dataOut3DArray[i][j][c] = exp_val;
                }
            }
        }

        for (std::size_t i = 0; i < outHeight; ++i) {
            for (std::size_t j = 0; j < outWidth; ++j) {
                float sum_exp = 0.0;
                for (std::size_t c = 0; c < outColorChannels; ++c) {
                    sum_exp += dataOut3DArray[i][j][c];
                }

                for (std::size_t c = 0; c < outColorChannels; ++c) {
                    dataOut3DArray[i][j][c] /= sum_exp;
                }
            }
        }

    }

    // Compute the convolution using SIMD
    void SoftmaxLayer::computeSIMD(const LayerData& dataIn) const {
        // TODO: Your Code Here...
    }
};  // namespace ML