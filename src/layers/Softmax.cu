#include "hip/hip_runtime.h"
#include "Softmax.h"

#include <iostream>
#include <cmath>

#include "../Types.h"
#include "../Utils.h"
#include "Layer.h"


namespace ML {

    void SoftmaxLayer::computeNaive(const LayerData& dataIn) const {
        Array1D_fp32 dataInArray = dataIn.getData<Array1D_fp32>();
        std::size_t dataInSize = getInputParams().dims[0];

        Array1D_fp32 dataOutArray = outData.getData<Array1D_fp32>();
        std::size_t dataOutSize = getOutputParams().dims[0];

        if (dataInSize != dataOutSize) {
            throw std::invalid_argument("Input and output sizes do not match.");
        }

        auto maxInput = std::numeric_limits<fp32>::min();
        for (std::size_t i = 0; i < dataInSize; ++i) {
            if (dataInArray[i] > maxInput) {
                maxInput = dataInArray[i];
            }
        }

        auto sumExp = 0.0f;
        for (std::size_t i = 0; i < dataInSize; ++i) {
            dataOutArray[i] = std::exp(dataInArray[i] - maxInput);
            sumExp += dataOutArray[i];
        }

        for (std::size_t i = 0; i < dataInSize; ++i) {
            dataOutArray[i] /= sumExp;
        }
    }

    // Compute the convolution using SIMD
    void SoftmaxLayer::computeSIMD(const LayerData& dataIn) const {
        // TODO: Your Code Here...
    }
};  // namespace ML