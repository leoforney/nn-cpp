#include "hip/hip_runtime.h"
#include "Convolutional.h"

#include <iostream>
#include <thread>
#include <vector>
#include "../Types.h"
#include "../Utils.h"
#include "Layer.h"

__global__ void computeSimdKernel(float* dataIn,
                                  float* dataOut,
                                  float* weightData,
                                  std::size_t numFilters,
                                  std::size_t outHeight,
                                  std::size_t outWidth,
                                  std::size_t filterHeight,
                                  std::size_t filterWidth,
                                  std::size_t inputColorChannels,
                                  std::size_t inputHeight,
                                  std::size_t inputWidth) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int f = index / (outHeight * outWidth);
    index %= outHeight * outWidth;
    int y = index / outWidth;
    int x = index % outWidth;

    if (f < numFilters && y < outHeight && x < outWidth) {
        float sum = 0.0f;
        for (std::size_t dy = 0; dy < filterHeight; dy++) {
            for (std::size_t dx = 0; dx < filterWidth; dx++) {
                for (std::size_t c = 0; c < inputColorChannels; c++) {
                    std::size_t in_y = y + dy;
                    std::size_t in_x = x + dx;
                    if (in_y < inputHeight && in_x < inputWidth) {
                        sum += dataIn[(in_y * inputWidth + in_x) * inputColorChannels + c] * weightData[(dy * filterWidth + dx) * numFilters + f];
                    }
                }
            }
        }
        dataOut[(y * outWidth + x) * numFilters + f] = sum;
    }
}

namespace ML {

void ConvolutionalLayer::computeNaive(const LayerData& dataIn) const {

    Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
    std::size_t inputHeight = getInputParams().dims[0];
    std::size_t inputWidth = getInputParams().dims[1];
    std::size_t inputColorChannels = getInputParams().dims[2];

    Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();
    auto outDims = getOutputParams();
    std::size_t outHeight = outDims.dims[0];
    std::size_t outWidth = outDims.dims[1];
    std::size_t outColorChannels = outDims.dims[2];

    std::size_t filterHeight = weightParam.dims[0];
    std::size_t filterWidth = weightParam.dims[1];
    std::size_t numFilters = weightParam.dims[3];

    Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

    for (std::size_t out_y = 0; out_y < outHeight; out_y++) {
        for (std::size_t out_x = 0; out_x < outWidth; out_x++) {
            for (std::size_t f = 0; f < numFilters; f++) {
                float sum = 0.0f;
                for (std::size_t dy = 0; dy < filterHeight; dy++) {
                    for (std::size_t dx = 0; dx < filterWidth; dx++) {
                        for (std::size_t c = 0; c < inputColorChannels; c++) {
                            std::size_t in_y = out_y + dy;
                            std::size_t in_x = out_x + dx;
                            if (in_y < inputHeight && in_x < inputWidth) {
                                sum += dataIn3DArray[in_y][in_x][c] * weightData4DArray[dy][dx][c][f];
                            }
                        }
                    }
                }
                dataOut3DArray[out_y][out_x][f] = sum;
            }
        }
    }
}

void ConvolutionalLayer::computeThreaded(const LayerData& dataIn) const {
    Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
    std::size_t inputHeight = getInputParams().dims[0];
    std::size_t inputWidth = getInputParams().dims[1];
    std::size_t inputColorChannels = getInputParams().dims[2];

    Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();

    auto outDims = getOutputParams();
    std::size_t outHeight = outDims.dims[0];
    std::size_t outWidth = outDims.dims[1];
    std::size_t outColorChannels = outDims.dims[2];

    std::size_t filterHeight = weightParam.dims[0];
    std::size_t filterWidth = weightParam.dims[1];
    std::size_t numFilters = weightParam.dims[3];

    Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

    std::vector<std::thread> threads;
    for (std::size_t f = 0; f < numFilters; f++) {
        threads.push_back(std::thread([=, &dataIn3DArray, &dataOut3DArray, &weightData4DArray]() {
            for (std::size_t out_y = 0; out_y < outHeight; out_y++) {
                for (std::size_t out_x = 0; out_x < outWidth; out_x++) {
                    float sum = 0.0f;
                    for (std::size_t dy = 0; dy < filterHeight; dy++) {
                        for (std::size_t dx = 0; dx < filterWidth; dx++) {
                            for (std::size_t c = 0; c < inputColorChannels; c++) {
                                std::size_t in_y = out_y + dy;
                                std::size_t in_x = out_x + dx;
                                if (in_y < inputHeight && in_x < inputWidth) {
                                    sum += dataIn3DArray[in_y][in_x][c] * weightData4DArray[dy][dx][c][f];
                                }
                            }
                        }
                    }
                    dataOut3DArray[out_y][out_x][f] = sum;
                }
            }
        }));
    }
    for (auto& thread : threads) thread.join();
}

void ConvolutionalLayer::computeTiled(const LayerData& dataIn) const {
    static const std::size_t tileSize = 8;

    Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
    std::size_t inputHeight = getInputParams().dims[0];
    std::size_t inputWidth = getInputParams().dims[1];
    std::size_t inputColorChannels = getInputParams().dims[2];

    Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();

    auto outDims = getOutputParams();
    std::size_t outHeight = outDims.dims[0];
    std::size_t outWidth = outDims.dims[1];
    std::size_t outColorChannels = outDims.dims[2];

    std::size_t filterHeight = weightParam.dims[0];
    std::size_t filterWidth = weightParam.dims[1];
    std::size_t numFilters = weightParam.dims[3];

    Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

    for (std::size_t f = 0; f < numFilters; f++) {
        for (std::size_t out_y_tile = 0; out_y_tile < outHeight; out_y_tile += tileSize) {
            for (std::size_t out_x_tile = 0; out_x_tile < outWidth; out_x_tile += tileSize) {
                for (std::size_t y = out_y_tile; y < std::min(out_y_tile + tileSize, outHeight); y++) {
                    for (std::size_t x = out_x_tile; x < std::min(out_x_tile + tileSize, outWidth); x++) {
                        float sum = 0.0f;
                        for (std::size_t dy = 0; dy < filterHeight; dy++) {
                            for (std::size_t dx = 0; dx < filterWidth; dx++) {
                                for (std::size_t c = 0; c < inputColorChannels; c++) {
                                    std::size_t in_y = y + dy;
                                    std::size_t in_x = x + dx;
                                    if (in_y < inputHeight && in_x < inputWidth) {
                                        sum += dataIn3DArray[in_y][in_x][c] * weightData4DArray[dy][dx][c][f];
                                    }
                                }
                            }
                        }
                        dataOut3DArray[y][x][f] = sum;
                    }
                }
            }
        }
    }
}

    void ConvolutionalLayer::computeSIMD(const LayerData& dataIn) const {
        Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
        std::size_t inputHeight = getInputParams().dims[0];
        std::size_t inputWidth = getInputParams().dims[1];
        std::size_t inputColorChannels = getInputParams().dims[2];

        Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();
        auto outDims = getOutputParams();
        std::size_t outHeight = outDims.dims[0];
        std::size_t outWidth = outDims.dims[1];
        std::size_t outColorChannels = outDims.dims[2];

        std::size_t filterHeight = weightParam.dims[0];
        std::size_t filterWidth = weightParam.dims[1];
        std::size_t numFilters = weightParam.dims[3];

        Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

        dim3 threadsPerBlock(256);
        dim3 numBlocks((outHeight * outWidth * numFilters + threadsPerBlock.x - 1) / threadsPerBlock.x);

        float* d_dataIn;
        float* d_dataOut;
        float* d_weightData;
        hipMalloc(&d_dataIn, inputHeight * inputWidth * inputColorChannels * sizeof(float));
        hipMalloc(&d_dataOut, outHeight * outWidth * outColorChannels * sizeof(float));
        hipMalloc(&d_weightData, filterHeight * filterWidth * numFilters * sizeof(float));

        hipMemcpy(d_dataIn, &dataIn3DArray[0][0][0], inputHeight * inputWidth * inputColorChannels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_weightData, &weightData4DArray[0][0][0][0], filterHeight * filterWidth * numFilters * sizeof(float), hipMemcpyHostToDevice);

        computeSimdKernel<<<numBlocks, threadsPerBlock>>>(d_dataIn, d_dataOut, d_weightData, numFilters, outHeight, outWidth, filterHeight, filterWidth, inputColorChannels, inputHeight, inputWidth);

        hipMemcpy(&dataOut3DArray[0][0][0], d_dataOut, outHeight * outWidth * numFilters * sizeof(float), hipMemcpyDeviceToHost);

        hipFree(d_dataIn);
        hipFree(d_dataOut);
        hipFree(d_weightData);
    }
};  // namespace ML