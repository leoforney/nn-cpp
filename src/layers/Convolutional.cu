#include "hip/hip_runtime.h"
#include "Convolutional.h"

#include <iostream>
#include <thread>
#include <vector>
#include "../Types.h"
#include "../Utils.h"
#include "Layer.h"

__global__ void computeSimdKernel(float* dataIn,
                                  float* dataOut,
                                  float* weightData,
                                  std::size_t numFilters,
                                  std::size_t outHeight,
                                  std::size_t outWidth,
                                  std::size_t filterHeight,
                                  std::size_t filterWidth,
                                  std::size_t inputColorChannels,
                                  std::size_t inputHeight,
                                  std::size_t inputWidth) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int f = index / (outHeight * outWidth);
    index %= outHeight * outWidth;
    int y = index / outWidth;
    int x = index % outWidth;

    if (f < numFilters && y < outHeight && x < outWidth) {
        float sum = 0.0f;
        for (std::size_t dy = 0; dy < filterHeight; dy++) {
            for (std::size_t dx = 0; dx < filterWidth; dx++) {
                for (std::size_t c = 0; c < inputColorChannels; c++) {
                    std::size_t in_y = y + dy;
                    std::size_t in_x = x + dx;
                    if (in_y < inputHeight && in_x < inputWidth) {
                        sum += dataIn[(in_y * inputWidth + in_x) * inputColorChannels + c] * weightData[(dy * filterWidth + dx) * numFilters + f];
                    }
                }
            }
        }
        dataOut[(y * outWidth + x) * numFilters + f] = sum;
    }
}

namespace ML {

    void ConvolutionalLayer::computeNaive(const LayerData& dataIn) const {

        Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
        std::size_t inputHeight = getInputParams().dims[0];
        std::size_t inputWidth = getInputParams().dims[1];
        std::size_t inputColorChannels = getInputParams().dims[2];

        Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();
        auto outDims = getOutputParams();
        std::size_t outHeight = outDims.dims[0];
        std::size_t outWidth = outDims.dims[1];
        std::size_t outColorChannels = outDims.dims[2];

        std::size_t filterHeight = weightParam.dims[0];
        std::size_t filterWidth = weightParam.dims[1];
        std::size_t numFilters = weightParam.dims[3];

        Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

        for (std::size_t out_y = 0; out_y < outHeight; out_y++) {
            for (std::size_t out_x = 0; out_x < outWidth; out_x++) {
                for (std::size_t f = 0; f < numFilters; f++) {
                    float sum = 0.0f;
                    for (std::size_t dy = 0; dy < filterHeight; dy++) {
                        for (std::size_t dx = 0; dx < filterWidth; dx++) {
                            for (std::size_t c = 0; c < inputColorChannels; c++) {
                                std::size_t in_y = out_y + dy;
                                std::size_t in_x = out_x + dx;
                                if (in_y < inputHeight && in_x < inputWidth) {
                                    sum += dataIn3DArray[in_y][in_x][c] * weightData4DArray[dy][dx][c][f];
                                }
                            }
                        }
                    }
                    dataOut3DArray[out_y][out_x][f] = sum;
                }
            }
        }
    }

    void ConvolutionalLayer::computeSIMD(const LayerData& dataIn) const {
        Array3D_fp32 dataIn3DArray = dataIn.getData<Array3D_fp32>();
        std::size_t inputHeight = getInputParams().dims[0];
        std::size_t inputWidth = getInputParams().dims[1];
        std::size_t inputColorChannels = getInputParams().dims[2];

        Array3D_fp32 dataOut3DArray = outData.getData<Array3D_fp32>();
        auto outDims = getOutputParams();
        std::size_t outHeight = outDims.dims[0];
        std::size_t outWidth = outDims.dims[1];
        std::size_t outColorChannels = outDims.dims[2];

        std::size_t filterHeight = weightParam.dims[0];
        std::size_t filterWidth = weightParam.dims[1];
        std::size_t numFilters = weightParam.dims[3];

        Array4D<fp32> weightData4DArray = weightData.getData<Array4D<fp32>>();

        dim3 threadsPerBlock(256);
        dim3 numBlocks((outHeight * outWidth * numFilters + threadsPerBlock.x - 1) / threadsPerBlock.x);

        float* d_dataIn;
        float* d_dataOut;
        float* d_weightData;
        hipMalloc(&d_dataIn, inputHeight * inputWidth * inputColorChannels * sizeof(float));
        hipMalloc(&d_dataOut, outHeight * outWidth * outColorChannels * sizeof(float));
        hipMalloc(&d_weightData, filterHeight * filterWidth * numFilters * sizeof(float));

        hipMemcpy(d_dataIn, &(dataIn3DArray[0][0][0]), inputHeight * inputWidth * inputColorChannels * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_weightData, &(weightData4DArray[0][0][0][0]), filterHeight * filterWidth * numFilters * sizeof(float), hipMemcpyHostToDevice);

        computeSimdKernel<<<numBlocks, threadsPerBlock>>>(d_dataIn, d_dataOut, d_weightData, numFilters, outHeight, outWidth, filterHeight, filterWidth, inputColorChannels, inputHeight, inputWidth);

        float *h_dataOut = new float[outHeight * outWidth * numFilters];
        hipMemcpy(h_dataOut, d_dataOut, outHeight * outWidth * numFilters * sizeof(float), hipMemcpyDeviceToHost);

        for (std::size_t y = 0; y < outHeight; y++) {
            for (std::size_t x = 0; x < outWidth; x++) {
                for (std::size_t f = 0; f < numFilters; f++) {
                    dataOut3DArray[y][x][f] = h_dataOut[(y * outWidth + x) * numFilters + f];
                }
            }
        }

        hipFree(d_dataIn);
        hipFree(d_dataOut);
        hipFree(d_weightData);

        delete[] h_dataOut;
    }
};  // namespace ML